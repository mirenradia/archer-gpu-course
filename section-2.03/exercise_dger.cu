/*
 * Introduction.
 *
 * An implementation of the blas level 2 routine dger(), which is
 * the operation
 *
 *   A_ij := A_ij + alpha x_i y_j
 *
 * where A is an m by n matrix, x is a vector of length m, y is
 * a vector of length n, and alpha is a constant. The data type
 * is double.
 *
 *
 * We will allocate a 1-dimensional object to handle the matrix
 * (with data type double) and address elements in the C-style
 * flattened order
 *
 *    A_ij (row i and column j) corresponds to data[i*ncol + j]
 *
 *
 * Copyright EPCC, The University of Edinburgh, 2023
 */

#include <assert.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__host__ void myErrorHandler(hipError_t ifail, const char * file,
                             int line, int fatal);

#define CUDA_ASSERT(call) { myErrorHandler((call), __FILE__, __LINE__, 1); }


/* Kernel parameters */

#define THREADS_PER_BLOCK_1D  256
#define THREADS_PER_BLOCK_2D   16

/* Kernel stub */

__global__ void myKernel(int mrow, int ncol, double alpha, double * x,
                         double * y, double * a) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  
  // if (i < mrow && j < ncol) {
    a[mrow*j + i] = a[mrow*j + i] + alpha*x[i]*y[j];
  // }

  return;
}

/* Main routine */

int main(int argc, char *argv[]) {

  int mrow = 1024;      /* Number of rows */
  int ncol =  512;      /* Number of columns */

  double alpha = 2.0;
  double * x = NULL;
  double * y = NULL;
  double * a = NULL;

  /* Check we have a GPU, and get device name from the cudaDeviceProp
   * structure. This is for information. */

  int ndevice = 0;
  int deviceNum = -1;
  hipDeviceProp_t prop;

  CUDA_ASSERT( hipGetDeviceCount(&ndevice) );

  if (ndevice == 0) {
     printf("No GPU available!\n");
     exit(0);
  }

  CUDA_ASSERT( hipGetDevice(&deviceNum) );
  CUDA_ASSERT( hipGetDeviceProperties(&prop, deviceNum) );
  printf("Device %d name: %s\n", deviceNum, prop.name);
  printf("Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);

  /* Establish host data (with some initial values for x and y) */

  // x = (double *) malloc(mrow*sizeof(double));
  // y = (double *) malloc(ncol*sizeof(double));
  // a = (double *) malloc(mrow*ncol*sizeof(double));
  // assert(x);
  // assert(y);
  // assert(a);
  CUDA_ASSERT( hipMallocManaged(&x, mrow*sizeof(double)) );
  CUDA_ASSERT( hipMallocManaged(&y, ncol*sizeof(double)) );
  CUDA_ASSERT( hipMallocManaged(&a, mrow*ncol*sizeof(double)) );

  for (int i = 0; i < mrow; i++) {
    x[i] = 1.0*i;
  }
  for (int j = 0; j < ncol; j++) {
    y[j] = 1.0*j;
    for (int i = 0; i < mrow; ++i) {
      a[j*mrow + i] = 0.0;
    }
  }

  /* Define the execution configuration and run the kernel */

  unsigned int nblockx = 1 + (mrow - 1)/THREADS_PER_BLOCK_2D;
  unsigned int nblocky = 1 + (ncol - 1)/THREADS_PER_BLOCK_2D;
  dim3 blocks = {nblockx, nblocky, 1};
  dim3 threadsPerBlock = {THREADS_PER_BLOCK_2D, THREADS_PER_BLOCK_2D, 1};

  myKernel<<<blocks, threadsPerBlock>>>(mrow, ncol, alpha, x, y, a);

  CUDA_ASSERT( hipPeekAtLastError() );
  CUDA_ASSERT( hipDeviceSynchronize() );


  int ncorrect = 0;
  printf("Results:\n");
  for (int i = 0; i < mrow; i++) {
    for (int j = 0; j < ncol; j++) {
      if (fabs(a[mrow*j + i] - alpha*x[i]*y[j]) < DBL_EPSILON) {
        ncorrect += 1;
      }
    }
  }
  printf("Number rows x cols %10d; correct: %10d\n", mrow*ncol, ncorrect);

  /* Release resources */

  CUDA_ASSERT( hipFree(y) );
  CUDA_ASSERT( hipFree(x) );
  CUDA_ASSERT( hipFree(a) );

  return 0;
}

/* It is important to check the return code from API calls, so the
 * follow function/macro allow this to be done concisely as
 *
 *   CUDA_ASSERT(cudaRunTimeAPIFunction(...));
 *
 * Return codes may be asynchronous, and thus misleading! */

__host__ void myErrorHandler(hipError_t ifail, const char * file,
                             int line, int fatal) {

  if (ifail != hipSuccess) {
    fprintf(stderr, "Line %d (%s): %s: %s\n", line, file,
            hipGetErrorName(ifail), hipGetErrorString(ifail));
    if (fatal) exit(ifail);
  }

  return;
}
